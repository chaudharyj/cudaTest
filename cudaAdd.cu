#include<iostream>
#include<hip/hip_runtime.h>
#include<cstdlib>
#include<ctime>

#define LIM 100

using namespace std;

__global__ void cudaAdd(int *d_a, int *d_b, int *d_c) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i<LIM) {
		d_c[i] = d_a[i] + d_b[i];
	}
}

int main() {

	int a[LIM],b[LIM],c[LIM];
	int *d_a, *d_b, *d_c;
	srand(time(NULL));
	for(int i = 0;i<LIM;i++) {
		a[i] = rand()%1000;
		b[i]= rand()%1000;
	}

	hipMalloc(&d_a,sizeof(int)*LIM);
	hipMalloc(&d_b,sizeof(int)*LIM);
	hipMalloc(&d_c,sizeof(int)*LIM);

	hipMemcpy(d_a,a,sizeof(int)*LIM,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,sizeof(int)*LIM,hipMemcpyHostToDevice);
	cudaAdd<<<(LIM/100)+1,100>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,sizeof(int)*LIM,hipMemcpyDeviceToHost);

	for(int i = 0;i<LIM;i++) {
		cout<<a[i]<<" + "<<b[i]<<" = "<<c[i]<<endl;
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	//delete[] a;
	//delete[] b;
	//delete[] c;
}
